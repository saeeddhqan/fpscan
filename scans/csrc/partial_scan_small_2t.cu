
#include <hip/hip_runtime.h>


template <typename tens>
__device__ void scan_forward_small_2t_load(
	const tens *__restrict__ Ax,
	tens * Bx,
	tens* partial_a,
	tens* partial_b,
	const uint tx,
	const uint offset,
	const uint chunk_offset
){
	partial_a[0] = tx == 0 ? (tens) 1.0 : Ax[offset];
	partial_b[0] = Bx[offset];
	tens gate = Ax[chunk_offset];
	partial_a[1] = partial_a[0] * gate;
	partial_b[1] = partial_b[0] * gate + Bx[chunk_offset];
}

template <typename tens, uint warps_per_block>
__global__  void scan_forward_small_2t(
	const tens *__restrict__ Ax,
	tens * Bx,
	const uint batch_stride,
	const uint dim_stride
) {
	const uint offset = blockIdx.x * batch_stride + blockIdx.y * dim_stride;
	const uint chunk_offset = offset + threadIdx.x * 2;
	const uint lane_id = threadIdx.x % 32;
	tens partial_a[2];
	tens partial_b[2];

	scan_forward_small_2t_load(Ax, Bx, partial_a, partial_b, threadIdx.x, offset, chunk_offset);
	
	#pragma unroll
	for (uint delta = 1; delta < 32; delta *= 2) {
		tens prev_gate = __shfl_up_sync(0xffffffff, partial_a[1], delta); // hardcoded step_thread - 1
		tens prev_token = __shfl_up_sync(0xffffffff, partial_b[1], delta); // hardcoded step_thread - 1

		if (lane_id >= delta) {
			partial_b[0] = prev_token * partial_a[0] + partial_b[0];
			partial_a[0] = prev_gate * partial_a[0];
			partial_b[1] = prev_token * partial_a[1] + partial_b[1];
			partial_a[1] = prev_gate * partial_a[1];
		}
	}

	Bx[chunk_offset] = partial_b[0];
	Bx[chunk_offset + 1] = partial_b[1];
}

