#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>

template <typename tens>
__global__ __forceinline__ __launch_bounds__(512, 32)
void scan(
	tens* __restrict__ A,
	tens* __restrict__ B
) {

	const unsigned int id = (blockIdx.x * 512) + threadIdx.x;
	const unsigned int lane_id = id % 32;
	tens value = B[id];
	tens gate = A[id];

	#pragma unroll
	for (unsigned int i = 1; i <= 32; i *= 2) {
		tens n = __shfl_up_sync(0xffffffff, value, i, 32);
		tens g = __shfl_up_sync(0xffffffff, gate, i, 32);
		if (lane_id >= i) {
			value += gate * n;
			gate *= g;
		}
	}

	B[id] = value;

}


template <typename tens, typename tens_t>
void myscan(const at::Tensor &Ax, const at::Tensor &Bx) {
	auto stream = at::cuda::getCurrentCUDAStream().stream();
	const auto sizes = Bx.sizes();
	const unsigned int batch = sizes[0];
	const auto strides = Bx.strides();
	const unsigned int batch_stride = strides[0];
	constexpr unsigned int block_size = 512;
	const unsigned int grid_size = (batch_stride * batch) / block_size;

	scan<tens><<<grid_size, block_size, 0, stream>>>(
		reinterpret_cast<tens*>(Ax.data_ptr<tens_t>()),
		reinterpret_cast<tens*>(Bx.data_ptr<tens_t>())
	);
	hipDeviceSynchronize();
}

at::Tensor myscan_forward(const at::Tensor &Ax, const at::Tensor &Bx) {

	if (Bx.scalar_type() == at::ScalarType::BFloat16) {
		myscan<__hip_bfloat16, at::BFloat16>(Ax, Bx);
	} else if (Bx.scalar_type() == at::ScalarType::Half) {
		myscan<__half, at::Half>(Ax, Bx);
	} else if (Bx.scalar_type() == at::ScalarType::Float) {
		myscan<float, float>(Ax, Bx);
	} else {
		TORCH_CHECK(false && "Invalid dtype");
	}

	return Bx;
}
