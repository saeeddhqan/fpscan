#include "hip/hip_runtime.h"
/*
We do not progress Ax for the next chunks...
We Only consider Bx of the previous chunk for the next chunk. I hope you succeed.
We use Bh, which is the hidden state of the previous layer to make k.
We use the current value to make q, and k.
*/

#include "shared.h"



template <typename tens>
__global__  void scan_kernel_large_4t_4096(
    const tens *__restrict__ Ax,
    tens *__restrict__ Bx,
    const tens *__restrict__ Bh,
    const tens *__restrict__ Wq,
    const tens *__restrict__ Wk,
    uint batch_stride, uint dim_stride
) {
    const uint warps_per_block = 32;
    const int bh_batch_stride = warps_per_block * gridDim.y;

    tens warp_q;
    __shared__ tens warp_k[warps_per_block];
    __shared__ tens warp_v[warps_per_block];
    __shared__ tens warp_r[warps_per_block];

    uint offset = (blockIdx.x * batch_stride + blockIdx.y * dim_stride) + threadIdx.x * 4;
    const uint warp_id = threadIdx.x >> 5;
    const uint lane_id = threadIdx.x & 31;
    const uint bh_offset = blockIdx.x * bh_batch_stride + blockIdx.y * warps_per_block + warp_id;

    tens partial_a[4];
    tens partial_b[4];

    partial_a[0] = threadIdx.x == 0 ? (tens) 1.0 : Ax[offset];
    partial_b[0] = Bx[offset];

    tens gate = Ax[offset + 1];
    partial_a[1] = partial_a[0] * gate;
    partial_b[1] = partial_b[0] * gate + Bx[offset + 1];

    gate = Ax[offset + 2];
    partial_a[2] = partial_a[1] * gate;
    partial_b[2] = partial_b[1] * gate + Bx[offset + 2];

    gate = Ax[offset + 3];
    partial_a[3] = partial_a[2] * gate;
    partial_b[3] = partial_b[2] * gate + Bx[offset + 3];

    #pragma unroll
    for (int delta = 1; delta < 32; delta *= 2) {
        tens prev_gate = __shfl_up_sync(0xffffffff, partial_a[3], delta);
        tens prev_token = __shfl_up_sync(0xffffffff, partial_b[3], delta);

        if (lane_id >= delta) {
            partial_b[0] = prev_token * partial_a[0] + partial_b[0];
            partial_a[0] = prev_gate * partial_a[0];
            partial_b[1] = prev_token * partial_a[1] + partial_b[1];
            partial_a[1] = prev_gate * partial_a[1];
            partial_b[2] = prev_token * partial_a[2] + partial_b[2];
            partial_a[2] = prev_gate * partial_a[2];
            partial_b[3] = prev_token * partial_a[3] + partial_b[3];
            partial_a[3] = prev_gate * partial_a[3];
        }
    }

    __syncwarp();

    if (lane_id == 31 && warp_id < warps_per_block - 1) {
        warp_k[warp_id] = Bh[bh_offset];
        warp_k[warp_id] = Wk[blockIdx.y] * warp_k[warp_id] + warp_k[warp_id];
        warp_v[warp_id] = partial_b[3];
        warp_q = Wq[blockIdx.y] * warp_v[warp_id] + warp_v[warp_id];
        warp_r[warp_id] = warp_v[warp_id];
    }

    __syncthreads();

    if (lane_id == 31 && warp_id && warp_id < warps_per_block - 1) {
        tens score = -1e10; 
        uint bid; // trouble
        #pragma unroll
        for (uint delta = 0; delta < warp_id; ++delta) {
            tens tmp_score = warp_k[delta] * warp_q;
            if (tmp_score > score) {
                score = tmp_score;
                bid = delta;
            }
        }
        warp_r[warp_id] = score * warp_v[bid] + warp_v[warp_id];
    }
    __syncthreads();

    if (warp_id > 0) {
        partial_b[0] = partial_b[0] + warp_r[warp_id - 1];
        partial_b[1] = partial_b[1] + warp_r[warp_id - 1];
        partial_b[2] = partial_b[2] + warp_r[warp_id - 1];
        partial_b[3] = partial_b[3] + warp_r[warp_id - 1];
    }
    Bx[offset] = partial_b[0];
    Bx[offset + 1] = partial_b[1];
    Bx[offset + 2] = partial_b[2];
    Bx[offset + 3] = partial_b[3];
}


template <typename tens, uint chunks_per_seq>
__global__  void scan_kernel_large_4t_32wpb(
    const tens *__restrict__ Ax,
    tens *__restrict__ Bx,
    const tens *__restrict__ Bh,
    const tens *__restrict__ Wq,
    const tens *__restrict__ Wk,
    uint batch_stride, uint dim_stride
) {
    const int steps_per_thread = 4;
    const int warps_per_block = 32;
    const int bh_dim_stride = warps_per_block * chunks_per_seq;
    const int bh_batch_stride = warps_per_block * chunks_per_seq * gridDim.y;

    tens warp_q;
    __shared__ tens warp_k[warps_per_block];
    __shared__ tens warp_v[warps_per_block];
    __shared__ tens warp_r[warps_per_block];
    __shared__ tens chunk_b;

    const uint seq_offset = blockIdx.x * batch_stride + blockIdx.y * dim_stride;
    const uint warp_id = threadIdx.x / 32;
    const uint lane_id = threadIdx.x % 32;
    const uint chunklen = blockDim.x * steps_per_thread; // thread * steps
    const uint bh_offset = blockIdx.x * bh_batch_stride + blockIdx.y * bh_dim_stride + warp_id;
    constexpr uint last_thread = steps_per_thread - 1;
    constexpr uint last_warp = 31;
    constexpr uint last_block = warps_per_block - 1;
    const tens empty_gate = 1.0; //constexpr?

    tens partial_a[steps_per_thread];
    tens partial_b[steps_per_thread];

    #pragma unroll
    for (uint chunk = 0; chunk < chunks_per_seq; chunk++) {
        const uint offset = seq_offset + chunk * chunklen;

        if (chunk) {
            __syncthreads();
        }

        #pragma unroll
        for (uint i = 0; i < steps_per_thread; ++i) {
            const uint chunk_offset = offset + (threadIdx.x * steps_per_thread + i);
            if (i == 0) {
                if (chunk == 0) {
                    partial_a[0] = threadIdx.x == 0 ? empty_gate : Ax[chunk_offset];
                    partial_b[0] = Bx[chunk_offset];
                } else {
                    if (threadIdx.x == 0) {
                        partial_b[0] = Ax[chunk_offset] * chunk_b + Bx[chunk_offset];
                    } else {
                        partial_a[0] = Ax[chunk_offset];
                        partial_b[0] = Bx[chunk_offset];
                    }
                }
            } else {
                tens gate = Ax[chunk_offset];
                partial_a[i] = partial_a[i - 1] * gate;
                partial_b[i] = partial_b[i - 1] * gate + Bx[chunk_offset];
            }
        }

        #pragma unroll
        for (uint delta = 1; delta < 32; delta *= 2) {
            tens prev_gate = __shfl_up_sync(0xffffffff, partial_a[last_thread], delta);
            tens prev_token = __shfl_up_sync(0xffffffff, partial_b[last_thread], delta);

            if (lane_id >= delta) {
                #pragma unroll
                for (uint i = 0; i < steps_per_thread; ++i) {
                    partial_b[i] = prev_token * partial_a[i] + partial_b[i];
                    partial_a[i] = prev_gate * partial_a[i];
                }
            }
        }

        __syncwarp();

        if (lane_id == 31 && warp_id < last_block) {
            warp_k[warp_id] = Bh[bh_offset];
            warp_k[warp_id] = Wk[blockIdx.y] * warp_k[warp_id] + warp_k[warp_id];
            warp_v[warp_id] = partial_b[last_thread];
            warp_q = Wq[blockIdx.y] * warp_v[warp_id] + warp_v[warp_id];
            warp_r[warp_id] = warp_v[warp_id];
        }

        __syncthreads();

        if (lane_id == 31 && warp_id && warp_id < last_block) {
            tens score = -1e10;
            uint bid; // big trouble if tmp_score got -inf
            #pragma unroll
            for (int delta = 0; delta < warp_id; ++delta) {
                tens tmp_score = warp_k[delta] * warp_q;
                if (tmp_score > score) {
                    score = tmp_score;
                    bid = delta;
                }
            }
            warp_r[warp_id] = score * warp_v[bid] + warp_v[warp_id];
        }
        __syncthreads();

        #pragma unroll
        for (uint i = 0; i < steps_per_thread; ++i) {
            if (warp_id > 0) {
                partial_b[i] = partial_b[i] + warp_r[warp_id - 1];
            }
            Bx[offset + threadIdx.x * steps_per_thread + i] = partial_b[i];
        }

        if (lane_id == last_warp && warp_id == last_block) {
            chunk_b = partial_b[last_thread];
        }
    }
}



template <typename tens, uint warps_per_block>
__global__  void scan_kernel_small_2t(
    const tens *__restrict__ Ax,
    tens *__restrict__ Bx,
    const tens *__restrict__ Bh,
    const tens *__restrict__ Wq,
    const tens *__restrict__ Wk,
    uint batch_stride, uint dim_stride
) {

    const int bh_batch_stride = warps_per_block * gridDim.y;

    tens warp_q;
    __shared__ tens warp_k[warps_per_block];
    __shared__ tens warp_v[warps_per_block];
    __shared__ tens warp_r[warps_per_block];

    uint offset = (blockIdx.x * batch_stride + blockIdx.y * dim_stride) + threadIdx.x * 2;
    const uint warp_id = threadIdx.x >> 5;
    const uint lane_id = threadIdx.x & 31;
    const uint bh_offset = blockIdx.x * bh_batch_stride + blockIdx.y * warps_per_block + warp_id;

    tens partial_a[2];
    tens partial_b[2];

    partial_a[0] = threadIdx.x == 0 ? (tens) 1.0 : Ax[offset];
    partial_b[0] = Bx[offset];
    tens gate = Ax[offset + 1];
    partial_a[1] = partial_a[0] * gate;
    partial_b[1] = partial_b[0] * gate + Bx[offset + 1];
    
    #pragma unroll
    for (int delta = 1; delta < 32; delta *= 2) {
        tens prev_gate = __shfl_up_sync(0xffffffff, partial_a[1], delta);
        tens prev_token = __shfl_up_sync(0xffffffff, partial_b[1], delta);

        if (lane_id >= delta) {
            partial_b[0] = prev_token * partial_a[0] + partial_b[0];
            partial_a[0] = prev_gate * partial_a[0];
            partial_b[1] = prev_token * partial_a[1] + partial_b[1];
            partial_a[1] = prev_gate * partial_a[1];
        }
    }

    __syncwarp();

    if (lane_id == 31 && warp_id < warps_per_block - 1) {
        warp_k[warp_id] = Bh[bh_offset];
        warp_k[warp_id] = Wk[blockIdx.y] * warp_k[warp_id] + warp_k[warp_id];
        warp_v[warp_id] = partial_b[1];
        warp_q = Wq[blockIdx.y] * partial_b[1] + partial_b[1];
        warp_r[warp_id] = warp_v[warp_id];
    }

    __syncthreads();

    if (lane_id == 31 && warp_id && warp_id < warps_per_block - 1) {
        tens score = -1e10;
        uint bid; // make sure tmp_score doesn't get -inf
        #pragma unroll
        for (int delta = 0; delta < warp_id; ++delta) {
            tens tmp_score = warp_k[delta] * warp_q;
            if (tmp_score > score) {
                score = tmp_score;
                bid = delta;
            }
        }
        warp_r[warp_id] = score * warp_v[bid] + warp_v[warp_id];
    }
    __syncthreads();

    if (warp_id > 0) {
        partial_b[0] = partial_b[0] + warp_r[warp_id - 1];
        partial_b[1] = partial_b[1] + warp_r[warp_id - 1];
    }

    Bx[offset] = partial_b[0];
    Bx[offset + 1] = partial_b[1];

}




template<typename tens, uint warps_per_block>
__global__ void scan_kernel_small_1t(
    const tens *__restrict__ Ax,
    tens *__restrict__ Bx,
    const tens *__restrict__ Bh,
    const tens *__restrict__ Wq,
    const tens *__restrict__ Wk,
    uint batch_stride, uint dim_stride, uint bh_batch_stride, uint bh_dim_stride
    )
{
//     const int bh_batch_stride = warps_per_block * gridDim.y;

    __shared__ tens warp_k[warps_per_block];
    __shared__ tens warp_v[warps_per_block];
    __shared__ tens warp_r[warps_per_block];
    tens warp_q;

    const uint offset = (blockIdx.x * batch_stride + blockIdx.y * dim_stride) + threadIdx.x;
    const uint warp_id = threadIdx.x / 32; // x / 32
    const uint lane_id = threadIdx.x % 32; // x % 32
    const uint bh_offset = blockIdx.x * bh_batch_stride + blockIdx.y * bh_dim_stride + warp_id;
    tens partial_a = threadIdx.x == 0 ? (tens) 1.0 : Ax[offset];
    tens partial_b = Bx[offset];

    #pragma unroll
    for (int delta = 1; delta < 32; delta *= 2) {
        tens prev_gate = __shfl_up_sync(0xffffffff, partial_a, delta);
        tens prev_token = __shfl_up_sync(0xffffffff, partial_b, delta);

        if (lane_id >= delta) {
            partial_b = prev_token * partial_a + partial_b;
            partial_a = prev_gate * partial_a;
        }
    }

    __syncwarp();

    if (lane_id == 31 && warp_id < warps_per_block - 1) {
        warp_k[warp_id] = Bh[bh_offset];
        warp_k[warp_id] = Wk[blockIdx.y] * warp_k[warp_id] + warp_k[warp_id];
        warp_v[warp_id] = partial_b;
        warp_q = Wq[blockIdx.y] * partial_b + partial_b;
        // printf("%d: %fq, %fpb, %fw, %fk, %fv\n", warp_id, (float)warp_q, (float)partial_b, (float)Wq[blockIdx.y], (float)warp_k[warp_id], (float)warp_v[warp_id]);
        warp_r[warp_id] = partial_b;
    }

    __syncthreads();

    if (lane_id == 31 && warp_id > 0 && warp_id < warps_per_block - 1) {
        tens score = -1e10;
        int bid; // make sure tmp_score doesn't get -inf or inf, otherwise you are in a big trouble.  
        #pragma unroll
        for (int delta = 0; delta < warp_id; ++delta) {
            tens tmp_score = warp_k[delta] * warp_q;
            if (tmp_score > score) {
                score = tmp_score;
                bid = delta;
            }
        }
        warp_r[warp_id] = score * warp_v[bid] + warp_v[warp_id];
    }

    __syncthreads();

    if (warp_id > 0) {
        partial_b = partial_b + warp_r[warp_id - 1];
    }

    Bx[offset] = partial_b;
}




void dimwise_pscan(
    torch::Tensor &Ax,
    torch::Tensor &Bx,
    torch::Tensor &Bh,
    torch::Tensor &Wq,
    torch::Tensor &Wk)
{
    const auto strides = Bx.strides();
    const uint batch_stride = strides[0]; // maybe using block x, y, z reduces register pressure
    const uint dim_stride = strides[1];
    auto stream = at::cuda::getCurrentCUDAStream().stream();
    const auto sizes = Bx.sizes();
    const uint batch_size = sizes[0];
    const uint dim = sizes[1];
    const uint seqlen = sizes[2];
    const uint bh_batch_stride = Bh.strides()[0];
    const uint bh_dim_stride = Bh.strides()[1];

    dim3 grid(batch_size, dim);
    // torch::Tensor out = torch::empty({batch_size, dim, seqlen}, Bx.options());

    if (seqlen == 64) {
        DISPATCH_FLOAT_AND_HALF_AND_BF16(Ax.scalar_type(), Bx.scalar_type(),
            "dimwise scan",
            ([&]
                { scan_kernel_small_1t<input_t, 2><<<grid, 64, 2 * sizeof(input_t) * 3>>>(
                        static_cast<input_t *>(Ax.data_ptr()),
                        static_cast<input_t *>(Bx.data_ptr()),
                        static_cast<input_t *>(Bh.data_ptr()),
                        static_cast<input_t *>(Wq.data_ptr()),
                        static_cast<input_t *>(Wk.data_ptr()),
                        batch_stride, dim_stride, bh_batch_stride, bh_dim_stride
                    ); 
                }
            )
        );
    } else if (seqlen == 128) {
        DISPATCH_FLOAT_AND_HALF_AND_BF16(Ax.scalar_type(), Bx.scalar_type(),
            "dimwise scan",
            ([&]
                { scan_kernel_small_1t<input_t, 4><<<grid, 128, 4 * sizeof(input_t) * 3>>>(
                        static_cast<input_t *>(Ax.data_ptr()),
                        static_cast<input_t *>(Bx.data_ptr()),
                        static_cast<input_t *>(Bh.data_ptr()),
                        static_cast<input_t *>(Wq.data_ptr()),
                        static_cast<input_t *>(Wk.data_ptr()),
                        batch_stride, dim_stride, bh_batch_stride, bh_dim_stride
                    );
                }
            )
        );
    } else if (seqlen == 256) {
        DISPATCH_FLOAT_AND_HALF_AND_BF16(Ax.scalar_type(), Bx.scalar_type(),
            "dimwise scan",
            ([&]
                { scan_kernel_small_1t<input_t, 8><<<grid, 256, 8 * sizeof(input_t) * 3>>>(
                        static_cast<input_t *>(Ax.data_ptr()),
                        static_cast<input_t *>(Bx.data_ptr()),
                        static_cast<input_t *>(Bh.data_ptr()),
                        static_cast<input_t *>(Wq.data_ptr()),
                        static_cast<input_t *>(Wk.data_ptr()),
                        batch_stride, dim_stride, bh_batch_stride, bh_dim_stride
                    );
                }
            )
        );
    } else if (seqlen == 512) {
        DISPATCH_FLOAT_AND_HALF_AND_BF16(Ax.scalar_type(), Bx.scalar_type(),
            "dimwise scan",
            ([&]
                { scan_kernel_small_1t<input_t, 16><<<grid, 512, 16 * sizeof(input_t) * 3>>>(
                        static_cast<input_t *>(Ax.data_ptr()),
                        static_cast<input_t *>(Bx.data_ptr()),
                        static_cast<input_t *>(Bh.data_ptr()),
                        static_cast<input_t *>(Wq.data_ptr()),
                        static_cast<input_t *>(Wk.data_ptr()),
                        batch_stride, dim_stride, bh_batch_stride, bh_dim_stride
                    );
                }
            )
        );
    } else if (seqlen == 1024) {
        DISPATCH_FLOAT_AND_HALF_AND_BF16(Ax.scalar_type(), Bx.scalar_type(),
            "dimwise scan",
            ([&]
                { scan_kernel_small_2t<input_t, 16><<<grid, 512, 16 * sizeof(input_t) * 3>>>(
                        static_cast<input_t *>(Ax.data_ptr()),
                        static_cast<input_t *>(Bx.data_ptr()),
                        static_cast<input_t *>(Bh.data_ptr()),
                        static_cast<input_t *>(Wq.data_ptr()),
                        static_cast<input_t *>(Wk.data_ptr()),
                        batch_stride, dim_stride
                    );
                }
            )
        );
    } else if (seqlen == 2048) {
        DISPATCH_FLOAT_AND_HALF_AND_BF16(Ax.scalar_type(), Bx.scalar_type(),
            "dimwise scan",
            ([&]
                { scan_kernel_small_2t<input_t, 32><<<grid, 1024, 32 * sizeof(input_t) * 3>>>(
                        static_cast<input_t *>(Ax.data_ptr()),
                        static_cast<input_t *>(Bx.data_ptr()),
                        static_cast<input_t *>(Bh.data_ptr()),
                        static_cast<input_t *>(Wq.data_ptr()),
                        static_cast<input_t *>(Wk.data_ptr()),
                        batch_stride, dim_stride
                    );
                }
            )
        );
    }  else if (seqlen == 4096) {
        DISPATCH_FLOAT_AND_HALF_AND_BF16(Ax.scalar_type(), Bx.scalar_type(),
            "dimwise scan",
            ([&]
                { scan_kernel_large_4t_4096<input_t><<<grid, 1024, 32 * sizeof(input_t) * 3>>>(
                        static_cast<input_t *>(Ax.data_ptr()),
                        static_cast<input_t *>(Bx.data_ptr()),
                        static_cast<input_t *>(Bh.data_ptr()),
                        static_cast<input_t *>(Wq.data_ptr()),
                        static_cast<input_t *>(Wk.data_ptr()),
                        batch_stride, dim_stride
                    );
                }
            )
        );
    } else if (seqlen == 8192) {
        DISPATCH_FLOAT_AND_HALF_AND_BF16(Ax.scalar_type(), Bx.scalar_type(),
            "dimwise scan",
            ([&]
                { scan_kernel_large_4t_32wpb<input_t, 2><<<grid, 1024, 32 * sizeof(input_t) * 4>>>(
                        static_cast<input_t *>(Ax.data_ptr()),
                        static_cast<input_t *>(Bx.data_ptr()),
                        static_cast<input_t *>(Bh.data_ptr()),
                        static_cast<input_t *>(Wq.data_ptr()),
                        static_cast<input_t *>(Wk.data_ptr()),
                        batch_stride, dim_stride
                    );
                }
            )
        );
    } else if (seqlen == 16384) {
        DISPATCH_FLOAT_AND_HALF_AND_BF16(Ax.scalar_type(), Bx.scalar_type(),
            "dimwise scan",
            ([&]
                { scan_kernel_large_4t_32wpb<input_t, 4><<<grid, 1024, 32 * sizeof(input_t) * 4>>>(
                        static_cast<input_t *>(Ax.data_ptr()),
                        static_cast<input_t *>(Bx.data_ptr()),
                        static_cast<input_t *>(Bh.data_ptr()),
                        static_cast<input_t *>(Wq.data_ptr()),
                        static_cast<input_t *>(Wk.data_ptr()),
                        batch_stride, dim_stride
                    );
                }
            )
        );
    } else if (seqlen == 32768) {
        DISPATCH_FLOAT_AND_HALF_AND_BF16(Ax.scalar_type(), Bx.scalar_type(),
            "dimwise scan",
            ([&]
                { scan_kernel_large_4t_32wpb<input_t, 8><<<grid, 1024, 32 * sizeof(input_t) * 4>>>(
                        static_cast<input_t *>(Ax.data_ptr()),
                        static_cast<input_t *>(Bx.data_ptr()),
                        static_cast<input_t *>(Bh.data_ptr()),
                        static_cast<input_t *>(Wq.data_ptr()),
                        static_cast<input_t *>(Wk.data_ptr()),
                        batch_stride, dim_stride
                    );
                }
            )
        );
    } else if (seqlen == 65536) {
        DISPATCH_FLOAT_AND_HALF_AND_BF16(Ax.scalar_type(), Bx.scalar_type(),
            "dimwise scan",
            ([&]
                { scan_kernel_large_4t_32wpb<input_t, 16><<<grid, 1024, 32 * sizeof(input_t) * 4>>>(
                        static_cast<input_t *>(Ax.data_ptr()),
                        static_cast<input_t *>(Bx.data_ptr()),
                        static_cast<input_t *>(Bh.data_ptr()),
                        static_cast<input_t *>(Wq.data_ptr()),
                        static_cast<input_t *>(Wk.data_ptr()),
                        batch_stride, dim_stride
                    );
                }
            )
        );
    } else {
        TORCH_CHECK(false && "seqlen must be a power of 2, >= 32, <= 65536");
    }
}
